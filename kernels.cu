#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "cuwh.h"

using namespace std;

__device__ State state_add(const State &s1, const State &s2)
{
    State s;
    s.r = s1.r + s2.r;
    s.theta = s1.theta + s2.theta;
    s.phi = s1.phi + s2.phi;

    s.pr = s1.pr + s2.pr;
    s.ptheta = s1.ptheta + s2.ptheta;
    s.b = s1.b + s2.b;
    s.Bsq = s1.Bsq + s2.Bsq;
    return s;
}

__device__ State state_mul(double g, const State &s)
{
    State rs;
    rs.r = g*s.r;
    rs.theta = g*s.theta;
    rs.phi = g*s.phi;
    rs.pr = g*s.pr;
    rs.ptheta = g*s.ptheta;
    rs.b = g*s.b;
    rs.Bsq = g*s.Bsq;
    return rs;
}

__device__ double l(double r, double rhosq)
{
    
    return sqrt(rhosq + pow(r, 2));
}

// __device__ double l(double r, double a, double rho, double M)
// {
//     if (abs(r) < a) {
//         return rho;
//     }

//     double x = (2*(abs(r) - a)) / (pi*M);
//     return rho + M*(x*atan2(x) - log(1+pow(x,2))/2);

// }

__device__ double dldr(double r, double rhosq)
{
    return r/l(r, rhosq) ;
}

__device__ State rhs(const State &s, double rhosq)
{

    State ds;
    double rsq = pow(l(s.r, rhosq), 2);

    ds.r = s.pr;
    ds.theta = s.ptheta / rsq;
    ds.phi = s.b / (rsq*pow(sin(s.theta), 2));
    ds.pr = s.Bsq*(dldr(s.r, rhosq) / (pow(l(s.r, rhosq), 3)));
    ds.ptheta = (pow(s.b, 2)/rsq) * cos(s.theta)/pow(sin(s.theta), 3);
    ds.b = 0.0;
    ds.Bsq = 0.0;

    return ds;
}

__global__ void rk4_step(int N, State *states, double rhosq, double h)
{
    uint indx = blockIdx.x*blockDim.x + threadIdx.x;

    State s = states[indx];
    State y = s;

    State k1 = rhs(s, rhosq);
    s = state_add(y, state_mul(h/2, k1));
    State k2 = rhs(s, rhosq);
    s = state_add(y, state_mul(h/2, k2));
    State k3 = rhs(s, rhosq);
    s = state_add(y, state_mul(h, k3));
    State k4 = rhs(s, rhosq);

    s = state_add(y, state_mul(h/6, state_add(k4, state_add(state_mul(2.0, state_add(k2, k3)), k1))));
    states[indx] = s;
}

void curk4(int NA, int NB, int N, State *states, double rhosq, double h)
{
    rk4_step<<<NA, NB>>>(N, states, rhosq, h);
    hipDeviceSynchronize();
}


int compute_wh(State *ics, int Nx, int Ny, double rhosq)
{
    State *states_device;
    int N = Nx*Ny;
    hipError_t err = hipMalloc((void**) &states_device, N*sizeof(State));

    if (err != hipSuccess) {
        cout << "CUDA Memory Allocation Error" << endl;
        return -1;
    }

    err = hipMemcpy(states_device, &ics[0], N*sizeof(State), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cout << "CUDA Memory Copy Error" << endl;
        return -1;
    }

    int blockSize = 4096;
    int numBlocks = N / blockSize;

    // Integrate
    double dt = 1e-2;
    double t = 0.0;
    double tend = 100.0;
    int k = 0;
    while(t < tend)
    {
        curk4(blockSize, numBlocks, N, states_device, rhosq, dt);
        t += dt;
        k += 1;

        // if((k%1000) == 0)
        // {
        //     cout << "Time: " << k*dt << endl;
        // }
    }

    // Retrieve Results
    err = hipMemcpy(ics, states_device, N*sizeof(State), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        cout << "Retrieval error" << endl;
        return -1;
    }
    hipFree(states_device);
    return 0;

}